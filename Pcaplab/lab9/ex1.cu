#include"hip/hip_runtime.h"

#include<stdio.h>
#include<stdlib.h>

__global__ void rowadd(int *a,int *b,int* t){
	int tid=threadIdx.x,bid=blockIdx.x,width=gridDim.x;
	if(bid==0){
	for(int i=0;i<width;i++)
		t[tid*width+i]=a[tid*width+i]+b[tid*width+i];
		}
}

__global__ void coladd(int *a,int *b,int* t){
	int tid=threadIdx.x,bid=blockIdx.x,width=blockDim.x;
	if(bid==0){
	for(int i=0;i<width;i++)
		t[tid+width*i]=a[tid+width*i]+b[tid+width*i];
	}
}

__global__ void add(int *a,int *b,int* t){
	int tid=threadIdx.x,bid=blockIdx.x,width=blockDim.x;

	t[tid+width*bid]=a[tid+width*bid]+b[tid+width*bid];
		
}

int main(){
int *a,*t,*b,m,n,i,j;
int *d_a,*d_t,*d_b;
printf("Enter value of m: ");
scanf("%d",&m);
printf("Enter value of n: ");
scanf("%d",&n);
int size=sizeof(int)*m*n;
a=(int*)malloc(size);
b=(int*)malloc(size);
t=(int*)malloc(size);
printf("Enter input matrixA:\n");
for(i=0;i<m*n;i++)
	scanf("%d",&a[i]);
printf("Enter input matrixB:\n");
for(i=0;i<m*n;i++)
	scanf("%d",&b[i]);
hipMalloc((void**)&d_a,size);
hipMalloc((void**)&d_t,size);
hipMalloc((void**)&d_b,size);
hipMemcpy(d_a,a,size,hipMemcpyHostToDevice);
hipMemcpy(d_b,b,size,hipMemcpyHostToDevice);
hipMemcpy(d_t,t,size,hipMemcpyHostToDevice);
rowadd<<<n,m>>>(d_a,d_b,d_t);
hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
printf("Result:\n");
for(i=0;i<m;i++){
	for(j=0;j<n;j++)
		printf("%d\t",t[i*n+j]);
	printf("\n");
}

coladd<<<m,n>>>(d_a,d_b,d_t);
hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
printf("Result:\n");
for(i=0;i<m;i++){
	for(j=0;j<n;j++)
		printf("%d\t",t[i*n+j]);
	printf("\n");
}

add<<<m,n>>>(d_a,d_b,d_t);
hipMemcpy(t,d_t,size,hipMemcpyDeviceToHost);
printf("Result:\n");
for(i=0;i<m;i++){
	for(j=0;j<n;j++)
		printf("%d\t",t[i*n+j]);
	printf("\n");
}

hipFree(d_a);
hipFree(d_t);
return 0;
}