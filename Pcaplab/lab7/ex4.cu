#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add(float *a,float *c){
	int tid = threadIdx.x ;
	int bid=blockIdx.x;
	if(tid==1)
	c[bid]=sin(a[bid]);
}

int main(void){
	float a[100],c[100];
	float *d_a,*d_c;
	int size=sizeof(float)*100;
	int n;
	printf("Enter n:");
	scanf("%d",&n);
	printf("Enter arr A: ");
	for(int i=0;i<n;i++)
		scanf("%f",&a[i]);
	

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_c,size);

	

	hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
	

	add<<<n,256>>>(d_a,d_c);

	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
	for(int i=0;i<n;i++)
		printf("%f ",c[i]);
	hipFree(d_a);
	hipFree(d_c);
	return 0;
}