#include "hip/hip_runtime.h"

#include<stdio.h>

__global__ void add(int *a,int *b,int *c){
	int tid = threadIdx.x ;
	int bid=blockIdx.x;
	if(tid==1)
	c[bid]=a[bid]+b[bid];
}

int main(void){
	int a[100],b[100],c[100];
	int*d_a,*d_b,*d_c;
	int size=sizeof(int)*100;
	int n;
	printf("Enter n:");
	scanf("%d",&n);
	printf("Enter arr A: ");
	for(int i=0;i<n;i++)
		scanf("%d",&a[i]);
	printf("Enter arr B: ");
	for(int i=0;i<n;i++)
		scanf("%d",&b[i]);

	hipMalloc((void**)&d_a,size);
	hipMalloc((void**)&d_b,size);
	hipMalloc((void**)&d_c,size);

	

	hipMemcpy(d_a,&a,size,hipMemcpyHostToDevice);
	hipMemcpy(d_b,&b,size,hipMemcpyHostToDevice);

	add<<<n,256>>>(d_a,d_b,d_c);

	hipMemcpy(&c,d_c,size,hipMemcpyDeviceToHost);
	for(int i=0;i<n;i++)
		printf("%d ",c[i]);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	return 0;
}