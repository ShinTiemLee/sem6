#include"hip/hip_runtime.h"

#include<string.h>
#include<stdio.h>
#include<stdlib.h>

#define N 1024

__global__ void CUDACount(char* A,char* word,int *d_countA,int *d_countword,unsigned int *d_count){
	int i=threadIdx.x;
	if(i<=(*d_countA-*d_countword)){
		int flag=0;
		for(int j=0;j<*d_countword-1;j++){
			if(A[i]!=word[j]){
				flag=1;
				break;
			}
			i++;
		}
		if(flag==0)
			atomicAdd(d_count,1);
	}
}

int main(){
	char A[N],word[20];
	char *d_A,*d_word;
	unsigned int *count=0,*d_count,result;
	int *d_countA,*d_countword;
	printf("Enter a string: ");
	fgets(A,N,stdin);
	printf("Enter word to count: ");
	fgets(word,20,stdin);
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMalloc((void**)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void**)&d_word,strlen(word)*sizeof(char));
	hipMalloc((void**)&d_countA,sizeof(int));
	hipMalloc((void**)&d_countword,sizeof(int));
	hipMalloc((void**)&d_count,sizeof(unsigned int));
	int len1=strlen(A);
	int len2=strlen(word);
	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_word,word,strlen(word)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_countA,&len1,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_countword,&len2,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(unsigned int),hipMemcpyHostToDevice);

	hipError_t error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR1: %s\n",hipGetErrorString(error));
	}
	CUDACount<<<1,strlen(A)>>>(d_A,d_word,d_countA,d_countword,d_count);
	error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR2: %s\n",hipGetErrorString(error));
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipMemcpy(&result,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);
	printf("Total occurences of %s=%u",word,result);
	printf("TimeTaken=%f",elapsedTime);
	hipFree(d_A);
	hipFree(d_count);
	return 0;
}