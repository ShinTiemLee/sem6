#include"hip/hip_runtime.h"

#include<string.h>
#include<stdio.h>
#include<stdlib.h>

#define N 1024

__global__ void CUDACount(char* A,char* RS, int *d_countA){
	int i=threadIdx.x;
	int sum=0;
	for(int x=0;x<=i;x++)
		sum+=(*d_countA-1-x);
	for(int j=0;j<(*d_countA-i-1);j++){
		RS[sum+j]=A[j];
	}
}

int main(){
	char A[N],RS[N];
	char *d_A,*d_RS;
	int *d_countA;
	printf("Enter a string: ");
	fgets(A,N,stdin);
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	int len1=strlen(A);
	hipMalloc((void**)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void**)&d_RS,strlen(RS)*sizeof(char));
	hipMalloc((void**)&d_countA,sizeof(int));
	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_RS,RS,strlen(RS)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_countA,&len1,sizeof(int),hipMemcpyHostToDevice);
	

	hipError_t error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR1: %s\n",hipGetErrorString(error));
	}
	CUDACount<<<1,strlen(A)>>>(d_A,d_RS,d_countA);
	error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR2: %s\n",hipGetErrorString(error));
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipMemcpy(RS,d_RS,strlen(RS)*sizeof(char),hipMemcpyDeviceToHost);
	printf("RS = %s",RS);
	printf("TimeTaken=%f",elapsedTime);
	hipFree(d_A);
	
	return 0;
}