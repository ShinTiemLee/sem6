#include"hip/hip_runtime.h"

#include<string.h>
#include<stdio.h>
#include<stdlib.h>

#define N 1024

__global__ void CUDACount(char* A,unsigned int *d_count){
	int i=threadIdx.x;
	if(A[i]=='a')
		atomicAdd(d_count,1);
}

int main(){
	char A[N];
	char *d_A;
	unsigned int *count=0,*d_count,result;
	printf("Enter a string: ");
	fgets(A,N,stdin);
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	hipMalloc((void**)&d_A,strlen(A)*sizeof(char));
	hipMalloc((void**)&d_count,sizeof(unsigned int));
	hipMemcpy(d_A,A,strlen(A)*sizeof(char),hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(unsigned int),hipMemcpyHostToDevice);

	hipError_t error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR1: %s\n",hipGetErrorString(error));
	}
	CUDACount<<<1,strlen(A)>>>(d_A,d_count);
	error=hipGetLastError();
	if(error!=hipSuccess){
	printf("CUDA ERROR2: %s\n",hipGetErrorString(error));
	}
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime,start,stop);
	hipMemcpy(&result,d_count,sizeof(unsigned int),hipMemcpyDeviceToHost);
	printf("Total occurences of a=%u",result);
	printf("TimeTaken=%f",elapsedTime);
	hipFree(d_A);
	hipFree(d_count);
	return 0;
}